
#include <hip/hip_runtime.h>
#define BLUR_SIZE 1


__global__
void blurKernel(unsigned char* in, unsigned char* out, int w, int h) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    if (col < w && row < h) {
        int pixVal = 0;
        int pixels = 0;

        for (int blurRow=-BLUR_SIZE; blurRow<BLUR_SIZE+1; ++blurRow) {
            int curRow = row + blurRow;
            for (int blurCol=-BLUR_SIZE; blurCol<BLUR_SIZE+1; ++blurCol) {
                int curCol = col + blurCol;

                if (curRow>=0 && curRow<h && curCol>=0 && curCol<w) {
                    pixVal += in[curRow*w + curCol];
                    ++pixels;
                }
            }
        }
        out[row*w + col] = (unsigned char) (pixVal/pixels);
    }
}